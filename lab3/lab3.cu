#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>

using namespace std;

struct position{
    int x;
    int y;
} ;

struct vector{
    double x;
    double y;
    double z;
} ;

struct matrix{
    double dots[3][3];
} ;


__constant__ vector vec[32];
__constant__ matrix dots[32];

vector void_vec[32];
matrix void_matr[32];

void inverseMatr(matrix &matrix_vals) {
    double M1 =  (matrix_vals.dots[1][1] * matrix_vals.dots[2][2] - matrix_vals.dots[2][1] * matrix_vals.dots[1][2]);
    double M2 = -(matrix_vals.dots[1][0] * matrix_vals.dots[2][2] - matrix_vals.dots[2][0] * matrix_vals.dots[1][2]);
    double M3 =  (matrix_vals.dots[1][0] * matrix_vals.dots[2][1] - matrix_vals.dots[2][0] * matrix_vals.dots[1][1]);

    double M4 = -(matrix_vals.dots[0][1] * matrix_vals.dots[2][2] - matrix_vals.dots[2][1] * matrix_vals.dots[0][2]);
    double M5 =  (matrix_vals.dots[0][0] * matrix_vals.dots[2][2] - matrix_vals.dots[2][0] * matrix_vals.dots[0][2]);
    double M6 = -(matrix_vals.dots[0][0] * matrix_vals.dots[2][1] - matrix_vals.dots[2][0] * matrix_vals.dots[0][1]);

    double M7 =  (matrix_vals.dots[0][1] * matrix_vals.dots[1][2] - matrix_vals.dots[1][1] * matrix_vals.dots[0][2]);
    double M8 = -(matrix_vals.dots[0][0] * matrix_vals.dots[1][2] - matrix_vals.dots[1][0] * matrix_vals.dots[0][2]);
    double M9 =  (matrix_vals.dots[0][0] * matrix_vals.dots[1][1] - matrix_vals.dots[1][0] * matrix_vals.dots[0][1]);

    double minor[3][3] = {{M1, M4, M7},{M2, M5, M8},{M3, M6, M9}};

    double D = matrix_vals.dots[0][0] * M1 - matrix_vals.dots[0][1] * (-M2) + matrix_vals.dots[0][2] * M3;
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            matrix_vals.dots[i][j] = minor[i][j] / D;
        }
    }
}

__device__ double findPixel(uchar4* dot, int id_X) {
    double div[3]
    double ans = 0.0;
    double Ans[3];
    Ans[0] = 0;
    Ans[1] = 0;
    Ans[2] = 0;

    div[0] = dot->x - void_vec[id_X].x;
    div[1] = dot->y - void_vec[id_X].y;
    div[2] = dot->z - void_vec[id_X].z;


    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            Ans[i] += void_matr[id_X].dots[j][i] * div[j];
        }
    }

    for (int i = 0; i < 3; i++) {
        ans += div[i] * Ans[i];
    }
    return -ans;
}


__global__ void mahalanobis_kernel(uchar4* photo_dots, int width, int height, int nc) {
    int id_X = blockIdx.x * blockDim.x + threadIdx.x;
    int id_Y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset_X = gridDim.x * blockDim.x;
    int offset_Y = gridDim.y * blockDim.y;

    for (int row = id_Y; row < height; row += offset_Y) {
        for (int col = id_X; col < width; col += offset_X) {
            uchar4 dot = photo_dots[row * width + col];
            double max = findPixel(&dot, 0);
            int timed = 0;
            for (int i = 1; i < nc; i++) {
                double next = findPixel(&dot, i);
                if (next > max) {
                    max = next;
                    timed = i;
                }
            }
            photo_dots[row * width + col].width = (unsigned char)timed;
        }
    }
}

void start(uchar4* photo_dots, vector<vector<position>> &input_photo, int nc, int width) {
    vector<vector> vec_values.resize(32);
    vector<matrix> matrix_values.resize(32);

    for (int i = 0; i < nc; i++) {

        vec_values[i].x = 0;
        vec_values[i].y = 0;
        vec_values[i].z = 0;

        for (int j = 0; j < input_photo[i].size(); j++) {
            position point = input_photo[i][j];
            uchar4 dots = photo_dots[width * point.y + point.x];

            vec_values[i].x += dots.x;
            vec_values[i].y += dots.y;
            vec_values[i].z += dots.z;
        }

        vec_values[i].x = vec_values[i].x / input_photo[i].size();
        vec_values[i].y = vec_values[i].y / input_photo[i].size();
        vec_values[i].z = vec_values[i].z / input_photo[i].size();

        for (int f = 0; f < input_photo[i].size(); f++) {
            position point = input_photo[i][f];
            uchar4 dots = photo_dots[width * point.y * w + point.x];

            matrix timed;
            timed.dots[0][0] = (dots.x - vec_values[i].x) * (dots.x - vec_values[i].x);
            timed.dots[0][1] = (dots.x - vec_values[i].x) * (dots.y - vec_values[i].y);
            timed.dots[0][2] = (dots.x - vec_values[i].x) * (dots.z - vec_values[i].z);
            timed.dots[1][0] = (dots.y - vec_values[i].y) * (dots.x - vec_values[i].x);
            timed.dots[1][1] = (dots.y - vec_values[i].y) * (dots.y - vec_values[i].y);
            timed.dots[1][2] = (dots.y - vec_values[i].y) * (dots.z - vec_values[i].z);
            timed.dots[2][0] = (dots.z - vec_values[i].z) * (dots.x - vec_values[i].x);
            timed.dots[2][1] = (dots.z - vec_values[i].z) * (dots.y - vec_values[i].y);
            timed.dots[2][2] = (dots.z - vec_values[i].z) * (dots.z - vec_values[i].z);

            for (int k = 0; k < 3; k++) {
                for (int l = 0; l < 3; l++) {
                    matrix_values[i].dots[k][l] += timed.dots[k][l];
                }
            }
        }

        if (input_photo[i].size() > 1) {
            diff = (double)(input_photo[i].size() - 1);
            for (auto & k : matrix_values[i].dots) {
                for (double & l : k) {
                    l /= diff;
                }
            }
        }
    }

    for (int i = 0; i < nc; ++i) {
        inverseMatr(matrix_values[i]);
        void_vec[i] = vec_values[i];
        void_matr[i] = matrix_values[i];
    }
}

int main() {
    string output_file_name, input_file_name;
    int nc, width, height;

    cin >> input_file_name;
    cin >> output_file_name;
    cin >> nc;


    vector<vector<position>> input_photo.resize(nc);

    int class_pixels_number;
    for (int row = 0; row < nc; row++) {
        cin >> class_pixels;
        input_photo[row].resize(class_pixels_number);
        for (int column = 0; column < class_pixels_number; column++) {
            cin >> input_photo[row][column].x >> input_photo[row][column].y;
        }
    }

    FILE* input_file  = fopen(input_file_name.c_str(), "rb");
    FILE* output_file = fopen(output_file_name.c_str(), "wb");

    fread(&width, sizeof(int), 1, input_file);
    fread(&height, sizeof(int), 1, output_file);

    uchar4* photo_dots = (uchar4*)malloc(sizeof(uchar4) * width * height);

    fread(photo_dots, sizeof(uchar4), width * height, input_file);
    fclose(input_file);


    start(photo_dots, input_photo, nc, width);
    hipMemcpyToSymbol(HIP_SYMBOL(vec), void_vec, 32 * sizeof(vector));
    hipMemcpyToSymbol(HIP_SYMBOL(dots), void_matr, 32 * sizeof(matrix));

    uchar4* output_dots;
    hipMalloc(&output_dots, sizeof(uchar4) * width * height);
    hipMemcpy(output_dots, photo_dots, sizeof(uchar4) * width * height, hipMemcpyHostToDevice);



    mahalanobis_kernel<<<dim3(32, 32), dim3(32, 32)>>>(output_dots, width, height, nc);
    hipGetLastError();
    hipMemcpy(photo_dots, output_dots, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);

    hipFree(output_dots);

    fwrite(&width, sizeof(int), 1, output_file);
    fwrite(&height, sizeof(int), 1, output_file);
    fwrite(photo_dots, sizeof(uchar4), width * height, output_file);
    fclose(output_file);

    free(photo_dots);

    return 0;
}